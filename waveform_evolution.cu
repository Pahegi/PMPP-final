#include "hip/hip_runtime.h"
#include <thrust/sort.h>
#include <thrust/unique.h>

#include <algorithm>
#include <waveform_evolution.hpp>

void calculate_blocks_and_threads_xdim(int n, int *num_blocks, int *num_threads) {
	int max_threads_per_block, max_grid_dim_x;

	// Get device properties
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);	// Assume using device 0
	max_threads_per_block = prop.maxThreadsPerBlock;
	max_grid_dim_x = prop.maxGridSize[0];  // Maximum number of blocks in the x-dimension

	// Set num_threads based on a typical value or maximum supported by the device
	*num_threads = (max_threads_per_block < 512) ? max_threads_per_block : 512;

	// Calculate num_blocks
	*num_blocks = (n + *num_threads - 1) / *num_threads;  // ceil(n / num_threads)

	// Ensure num_blocks does not exceed the maximum grid size in x-dimension
	if (*num_blocks > max_grid_dim_x) {
		printf("Error: Too many blocks required (%d), exceeds device capability (%d).\n", *num_blocks, max_grid_dim_x);
		*num_blocks = max_grid_dim_x;  // Cap at the maximum grid size
	}
}

__global__ void evolve_operator_kernel(
	std::uint64_t const *device_wavefunction,
	std::uint64_t *wave_out,
	std::size_t num_ed,
	std::uint64_t const activation,
	std::uint64_t const deactivation) {
	// only calculate if idx is within bounds
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= num_ed) return;

	// calculate new wavefunction
	auto op_ad = activation ^ deactivation;
	auto wave = device_wavefunction[idx];
	auto check_deactivation = deactivation & wave;
	auto check_activation = activation & (~wave);
	auto op = ((check_activation == activation) & (check_deactivation == deactivation)) * op_ad;
	auto wave_new = op ^ wave;

	// insert new wavefunction, if wave_new != wave
	wave_out[idx + num_ed] = (wave_new != wave) ? wave_new : 0;
}

cuda::std::pair<pmpp::cuda_ptr<std::uint64_t[]>, std::size_t> evolve_operator(
	cuda::std::span<std::uint64_t const> device_wavefunction,
	std::uint64_t activation, std::uint64_t deactivation) {
	// create output array double the size of input
	// auto wave_out = pmpp::make_managed_cuda_array<std::uint64_t>(2 * device_wavefunction.size());
	// auto wave_out_span = cuda::std::span(wave_out.get(), 2 * device_wavefunction.size());
	std::uint64_t *wave_out_ptr;
	hipMallocManaged(&wave_out_ptr, 2 * device_wavefunction.size() * sizeof(std::uint64_t));
	hipMemset(wave_out_ptr, 0, 2 * device_wavefunction.size() * sizeof(std::uint64_t));
	auto wave_out_span = cuda::std::span(wave_out_ptr, 2 * device_wavefunction.size());

	// Copy the input array to the output array
	size_t num_ed = device_wavefunction.size();
	hipMemcpy(wave_out_span.data(), device_wavefunction.data(), device_wavefunction.size() * sizeof(uint64_t), hipMemcpyDeviceToDevice);

	// optimize number of blocks and threads
	int num_blocks, num_threads;
	calculate_blocks_and_threads_xdim(num_ed, &num_blocks, &num_threads);

	// launch kernel
	evolve_operator_kernel<<<num_blocks, num_threads>>>(
		device_wavefunction.data(),
		wave_out_span.data(),
		num_ed,
		activation,
		deactivation);

	hipDeviceSynchronize();

	// Sort and remove duplicates
	thrust::sort(wave_out_span.data(), wave_out_span.data() + 2 * num_ed);
	size_t *new_end = thrust::unique(wave_out_span.data(), wave_out_span.data() + 2 * num_ed);

	// Check if the first element is zero and adjust the return value
	size_t shift = (wave_out_span[0] == 0) ? 1 : 0;
	size_t num_ed_out = static_cast<std::size_t>(new_end - wave_out_span.data() - shift);
	return {pmpp::cuda_ptr<std::uint64_t[]>(wave_out_span.data() + shift), num_ed_out};
}

cuda::std::pair<pmpp::cuda_ptr<std::uint64_t[]>, std::size_t> evolve_ansatz(
	cuda::std::span<std::uint64_t const> device_wavefunction,
	cuda::std::span<std::uint64_t const> activations,
	cuda::std::span<std::uint64_t const> deactivations) {
	cuda::std::pair<pmpp::cuda_ptr<std::uint64_t[]>, std::size_t> result;
	auto wave_out = pmpp::make_managed_cuda_array<std::uint64_t>(device_wavefunction.size());
	auto wave_out_span = cuda::std::span(wave_out.get(), device_wavefunction.size());

	std::copy_n(device_wavefunction.data(), device_wavefunction.size(), wave_out_span.data());

	for (size_t i = 0; i < activations.size(); i++) {
		result = evolve_operator(wave_out_span, activations[i], deactivations[i]);
		wave_out_span = cuda::std::span(result.first.get(), result.second);
	}

	return result;
}
