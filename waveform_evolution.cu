#include "hip/hip_runtime.h"
#include <thrust/sort.h>
#include <thrust/unique.h>

#include <algorithm>
#include <waveform_evolution.hpp>

void calculate_blocks_and_threads_xdim(int n, int *num_blocks, int *num_threads) {
	int max_threads_per_block, max_grid_dim_x;

	// Get device properties
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);	// Assume using device 0
	max_threads_per_block = prop.maxThreadsPerBlock;
	max_grid_dim_x = prop.maxGridSize[0];  // Maximum number of blocks in the x-dimension

	// Set num_threads based on a typical value or maximum supported by the device
	*num_threads = (max_threads_per_block < 512) ? max_threads_per_block : 512;

	// Calculate num_blocks
	*num_blocks = (n + *num_threads - 1) / *num_threads;  // ceil(n / num_threads)

	// Ensure num_blocks does not exceed the maximum grid size in x-dimension
	if (*num_blocks > max_grid_dim_x) {
		printf("Error: Too many blocks required (%d), exceeds device capability (%d).\n", *num_blocks, max_grid_dim_x);
		*num_blocks = max_grid_dim_x;  // Cap at the maximum grid size
	}
}

// cuda kernel
__global__ void evolve_operator_kernel(
	std::uint64_t const *device_wavefunction,
	std::uint64_t *wave_out,
	std::size_t num_ed,
	std::uint64_t const activation,
	std::uint64_t const deactivation) {
	// only calculate if idx is within bounds
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= num_ed) return;

	// calculate new wavefunction
	auto op_ad = activation ^ deactivation;
	auto wave = device_wavefunction[idx];
	auto check_deactivation = deactivation & wave;
	auto check_activation = activation & (~wave);
	auto op = ((check_activation == activation) & (check_deactivation == deactivation)) * op_ad;
	auto wave_new = op ^ wave;

	// insert new wavefunction, if wave_new != wave
	wave_out[idx + num_ed] = (wave_new != wave) ? wave_new : 0;
}

cuda::std::pair<pmpp::cuda_ptr<std::uint64_t[]>, std::size_t> evolve_operator(
	cuda::std::span<std::uint64_t const> device_wavefunction,
	std::uint64_t activation, std::uint64_t deactivation) {
	// create output array double the size of input
	printf("evolve_operator: Creating Output Array\n");
	auto wave_out = pmpp::make_managed_cuda_array<std::uint64_t>(2 * size(device_wavefunction));
	auto wave_out_span = cuda::std::span(wave_out.get(), 2 * size(device_wavefunction));
	size_t num_ed = size(device_wavefunction);

	// copy wavefunctions to output array
	printf("evolve_operator: Copying Wavefunctions to output Array\n");
	std::copy_n(device_wavefunction.data(), size(device_wavefunction), wave_out_span.data());

	// optimize number of blocks and threads
	int num_blocks, num_threads;
	calculate_blocks_and_threads_xdim(num_ed, &num_blocks, &num_threads);
	printf("evolve_operator: Blocks: %d, Threads per Block: %d\n", num_blocks, num_threads);

	// launch kernel
	printf("evolve_operator: Launching Kernel\n");
	evolve_operator_kernel<<<num_blocks, num_threads>>>(
		device_wavefunction.data(),
		wave_out_span.data(),
		num_ed,
		activation,
		deactivation);
	hipDeviceSynchronize();

	// Sort and remove duplicates
	printf("evolve_operator: Sorting and Removing Duplicates\n");
	thrust::sort(wave_out_span.data(), wave_out_span.data() + 2 * num_ed);
	size_t *new_end = thrust::unique(wave_out_span.data(), wave_out_span.data() + 2 * num_ed);

	// Check if the first element is zero and adjust the return value
	size_t shift = (wave_out_span[0] == 0) ? 1 : 0;
	return {pmpp::cuda_ptr<std::uint64_t[]>(wave_out.get() + shift), static_cast<std::size_t>(new_end - wave_out_span.data() - shift)};
}

cuda::std::pair<pmpp::cuda_ptr<std::uint64_t[]>, std::size_t> evolve_ansatz(
	cuda::std::span<std::uint64_t const> device_wavefunction,
	cuda::std::span<std::uint64_t const> activations,
	cuda::std::span<std::uint64_t const> deactivations) {
	return {nullptr, 0};
}
